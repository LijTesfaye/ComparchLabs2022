#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void cuda_hello()
{
    printf("Hello world!");
}

int main()
{
    
    cuda_hello<<<1,1>>>();

    return 0;
}
