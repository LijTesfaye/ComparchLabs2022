#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void mult(int* a, int* b, int* c)
{
    *c = *a * *b;
}

int main()
{
    int a, b, c;
    int* d_a, * d_b, * d_c;
    int size = sizeof(int);

    // Allocate space for device copies of a, b, c​
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    a = 10;
    b = 3;

    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

    mult << <1, 1 >> > (d_a, d_b, d_c);

    // Copy result back to host​
    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

    printf("c = a x b = 10 x 3 = %d", c);

    // Cleanup​
    hipFree(d_a); hipFree(d_b); hipFree(d_c);

    return 0;
}