#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

#define N 10

void random_ints(int* a, int M)
{
    for (int i = 0; i < M; i++)
        a[i] = rand() % 10;
}

__global__ void mult(int *a, int *b, int *c)
{
    c[blockIdx.x] = a[blockIdx.x] * b[blockIdx.x];
}

int main()
{
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int size = N * sizeof(int);

    // Allocate space for device copies of a, b, c​
    a = (int*)malloc(size); random_ints(a, N);
    b = (int*)malloc(size); random_ints(b, N);
    c = (int*)malloc(size);

    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    
    mult<<<N,1>>>(d_a,d_b,d_c);

    // Copy result back to host​
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // Cleanup​
    hipFree(d_a); hipFree(d_b); hipFree(d_c);

    for (int i = 0; i < N; i++)
        printf("[%d]: %d x %d = %d\n", i, a[i], b[i], c[i]);

    return 0;
}