#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

#define N 3
#define M 5

void random_ints(int* a, int F)
{
    for (int i = 0; i < F; i++)
        a[i] = rand() % 10;
}

__global__ void mult(int* a, int* b, int* c, int MAX)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if(index < MAX)
        c[index] = a[index] * b[index];
}

int main()
{
    int* a, * b, * c;
    int* d_a, * d_b, * d_c;
    int size = M * N * sizeof(int);

    // Allocate space for device copies of a, b, c​
    a = (int*)malloc(size); random_ints(a, M * N);
    b = (int*)malloc(size); random_ints(b, M * N);
    c = (int*)malloc(size);

    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    mult <<<N, M>>>(d_a, d_b, d_c, M * N);

    // Copy result back to host​
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // Cleanup​
    hipFree(d_a); hipFree(d_b); hipFree(d_c);

    printf("c = \n");
    for (int i = 0; i < N * M; i++)
        printf("[%d]: %d x %d = %d\n", i, a[i], b[i], c[i]);

    return 0;
}