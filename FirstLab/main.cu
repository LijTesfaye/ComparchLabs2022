#include "hip/hip_runtime.h"


#include <iostream>
#include <time.h>
#include <cstdlib>

#define N 4*4		//number of blocks to be run
#define M 2
using namespace std;

//case of <<<1, 1>>> (1 block, with 1 thread)
__global__  void multiply1_1(int *a, int *b, int *result) {
    *result = *a * *b;
}


//case of <<<N, 1>>> (N blocks with 1 thread per block)
//using blockIdx.x to index into the passed array, we make each block handle
//a different element of the array
/*
	if N = 4 with a single thread for each block we will have

	Block 0 -> result[0] = a[0] + b[0]
	Block 1 -> result[1] = a[1] + b[1]
	Block 2 -> result[2] = a[2] + b[2]
	Block 3 -> result[3] = a[3] + b[3]
	executed in parallel on the GPU
*/
__global__ void multiplyN_1(int *a, int *b, int *result) {
    result[blockIdx.x] = a[blockIdx.x] * b[blockIdx.x];
}


/*

*/
__global__ void multiply(int *a, int *b, int *result, int n) {
    result[threadIdx.x] = a[threadIdx.x] * b[threadIdx.x];
    //printf("%d x %d = %d\n", a[threadIdx.x], b[threadIdx.x], result[threadIdx.x]);
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < n) result[index] = a[index] * b[index];
}




void random_ints(int *a, int n) {
    for (int i = 0; i < n; ++i)
        a[i] = rand() % 100 + 1;
}

void display(int *a, int *b, int *result, int n) {
    for (int i = 0; i < n; i++) {
        printf("%d  x  %d  =  %d\n", a[i], b[i], result[i]);
    }
}



int main(void) {
    int *a, *b, *result; // host copies of a, b, result
    int *d_a, *d_b, *d_result; // device copies of a, b, result
    int size = N * sizeof(int);


    // Allocate space for device copies of a, b, c
    /*
    cudaMalloc((void**)&d_a, size);
    cudaMalloc((void**)&d_b, size);
    cudaMalloc((void**)&d_result, size);
    */

    // Alloc space for host copies of a, b, c and setup input values
    a = (int*)malloc(size);
    random_ints(a, N);
    b = (int*)malloc(size);
    random_ints(b, N);
    result = (int*)malloc(size);

    // Alloc space for device copies of a, b, c
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_result, size);


    //setup variable values
    /*
    scanf("%d", &a);
    scanf("%d", &b);
    */


    // Copy inputs from host to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);


    //multiply1_1 <<< 1, 1>>> (d_a, d_b, d_result);

    //Launches multiplyN_1 on GPU with N blocks
    multiply <<<(N + M - 1) / M, M>>> (d_a, d_b, d_result, N);



    // Copy result back to host
    hipMemcpy(result, d_result, size, hipMemcpyDeviceToHost);



    //showing the result
    //printf("%d x %d = %d\n", a, b, result);

    display(a, b, result, N);



    // Cleanup
    free(a);
    free(b);
    free(result);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_result);
    return 0;
}