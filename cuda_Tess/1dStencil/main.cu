
#include "hip/hip_runtime.h"

#include <iostream>
//define vector length, stencil radius,
#define N (4*4)
#define RADIUS 3
#define GRIDSIZE 8 //N,M
#define BLOCKSIZE 32
//
int gridSize  = GRIDSIZE;
int blockSize = BLOCKSIZE;
/*
-------------------------------------------------------
CUDA device function that performs 1D stencil operation
-------------------------------------------------------
*/
__global__
void stencil_1D(int *in, int *out, int dim)
{
    __shared__ int temp[BLOCKSIZE + 2*RADIUS];

    int gindex = threadIdx.x + blockDim.x * blockIdx.x; //global index
    int stride = gridDim.x * blockDim.x;  //reason  on the slide.
    int tid = threadIdx.x;
    int lindex = threadIdx.x + RADIUS; //shared memory local index

    // Go through all data
    // Step all threads in a block to avoid synchronization problem
    while ( gindex < dim + blockDim.x)
    {
        if(gindex < dim)
        {
            temp[lindex] = in[gindex];
        }
        else
        {
            temp[lindex] = 0;
        }
        if(tid < RADIUS)
        {
            if(gindex < RADIUS)
            {
                temp[lindex - RADIUS] = 0;
            }
            else
            {
                temp[lindex - RADIUS] = in[gindex - RADIUS];
            }
            if(gindex + BLOCKSIZE >= dim)
            {
                temp[lindex + BLOCKSIZE] = 0;
            }
            else
            {
                temp[lindex + BLOCKSIZE] = in[gindex + BLOCKSIZE];
            }
        }
        __syncthreads(); // this is for the synchronization of the threads, avoi the race condition.

        // Apply the stencil
        int result = 0;
        for (int offset = -RADIUS; offset <= RADIUS; offset++) {
            if ( lindex + offset < dim && lindex + offset > -1)
                result += temp[lindex + offset];
        }
        // Store the result
        if (gindex < dim)
            out[gindex] = result;

        // Update global index and quit if we are done
        gindex += stride;
        __syncthreads(); // avoid the race condition
    }
}
//displays the results of the 1D stencil program
void display(int *a,int n) {
    for (int i = 0; i < n; i++) {
        printf("out is :%d", a[i]);
        printf("\n");
    }
}
//
// this function assigns a random integer to the input variable
void random_ints(int *a, int n) {
    for (int i = 0; i < n; ++i)
        a[i] = rand() % 100 + 1;
}
/*
------------
main program
------------
*/
int main(void) {

    int *in, *out;
    int *d_in, *d_out;
    int size = N * sizeof(int);

    //allocate a pace to the host memory
    in = (int *) malloc(size);
    random_ints(in, N);
    out = (int *) malloc(size);
    random_ints(out, N);

    // allocate device memory
    hipMalloc((void **) &d_in, size);
    hipMalloc((void **) &d_out, size);

    // copy input data to device
    hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);

    //----------------------------------------------------------
    // CODE TO RUN AND TIME THE STENCIL KERNEL.
    //----------------------------------------------------------
    stencil_1D<<<gridSize, blockSize>>>(d_in, d_out, N);
    //----------------------------------------------------------

    //copy the result from the Device to the Host
    //cudaMemcpy(c,d_c,size,cudaMemcpyDeviceToHost);
    hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);

    //display the results
    display(out, N);
    //
    // deallocate device memory
    hipFree(d_in);
    hipFree(d_out);
    // deallocate host memory
    free(in);
    free(out);
    return 0;
}