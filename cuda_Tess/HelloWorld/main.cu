#include "hip/hip_runtime.h"

#include <iostream>
#include <cstdlib>
//
// Learn these names by heart.
// the device is the name for the GPU
// the Host is the name for the CPU
//the __global__ command is a manifestation od a cuda C++ in your code. Hence, the code runs in the Device( i.e the GPU)
//
__global__
void hello_world(void ){
    printf("This is Hello world from Kernel");

}
// The saddest thing is that the Kernel is going to do nothing, which is annoying for real.But in the next part
// i.e adding numbers will be a bit better.
int main(void) {
    hello_world<<<1,1>>>(); // This is the kernel call.
    printf("This is Hello World from Main ! \n"); // this part of the code is displayed on the screen
    return 0;
}
