#include "hip/hip_runtime.h"

#include <iostream>

__global__
void threads_and_blocks(int *a, int *b,  int *c){
    int index=threadIdx.x+blockIdx.x*blockDim.x; // the index for a particular thread.
    /* threadId.x is the id of the thread
     * blockId.x is the id of that particular block
     * blockDim is the number of threads per a block, sometimes is referred to as M
     */
    c[index]=a[index]+b[index];
}
//
void random_ints(int *a, int n) {
    for (int i = 0; i < n; ++i)
        a[i] = rand() % 100 + 1;
}
//
void display(int *a, int *b, int *result, int n) {
    for (int i = 0; i < n; i++) {
        printf("%d  +  %d  =  %d\n", a[i], b[i], result[i]);
    }
}
//
#define N   (4*4)
#define M   2
//
int main() {
    //these are the host copies
    int *a, *b, *c ;
    // these are the device copies
    int *d_a , *d_b , *d_c;
    //the size is required
    int size=N*sizeof(int);
    //allocate space for the device copies
    hipMalloc((void **)&d_a,size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c,size);
    //allocate space for the host copies too
    a=(int *) malloc(size);
    random_ints(a,N);
    b=(int *) malloc(size);
    random_ints(b,N);
    c=(int *) malloc(size);
    //copy the variable to device
    hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
    hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);
    //launch the kernel
    threads_and_blocks<<<N/M,M>>>(d_a,d_b,d_c);
    //copy the results back to the Host
    hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);
    //display the results
    display(a,b,c,N);
    //clean up
        //the Device variables
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);
            // for the Host variables
            free(a);
            free(b);
            free(c);

    return 0;
}
