//for a cuda to run these the following two must be included.
#include "hip/hip_runtime.h"

//#include <iostream> // for input output
//
__global__
void add_two_num(int *a, int *b, int *c){
    *c=*a + *b;
}
void disp(int a, int b,int c){
    printf("%d + %d= %d",a,b,c);
}
// still this code is not parallel, but it is better than the Helloworld example in the previous  one.
// In this part ypu will understand the procedures to
//1. allocate space for the Device variables
//1.0 Specify the size tha type that you want e.g. size of int is used in this example;
//2. copy from  host
//3. copy from devices
//4.launch the kernel
//5. clean up the memory space allocated in step 1
//
int main() {
    //host copies of the numbers
    int a, b,c;
    //the copies of the Device
    int *d_a,*d_b,*d_c;
    //size specification
    int size=sizeof (int );
    //allocate space for device copies
    hipMalloc((void **) &d_a, size );
    hipMalloc((void **) &d_b, size);
    hipMalloc((void **) &d_c, size);

    //accept the values from the user
    printf("Enter the values of a: ");
    scanf("%d",&a);
    printf("Enter the values of b: ");
    //strtol('%d',&b);
    scanf("%d",&b);

    //copy the values to the Device
    hipMemcpy(d_a,&a,size,hipMemcpyHostToDevice);
    hipMemcpy(d_b,&b,size,hipMemcpyHostToDevice);

    //launch the kernel
    add_two_num<<<1,1>>>(d_a,d_b,d_c); //it is executing the kernel 1 times.
    //In the upcoming exercise we will see the vector addition for executing the kernel multiple times.
    //copy the result to the Host
    hipMemcpy(&c,d_c,size,hipMemcpyDeviceToHost);
    //call the display function to show the results
    disp(a,b,c);
    //cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}
