/* holds the cuda memory management functions like cudaMemcpy,
 cudaFree,cudaMemcpyHostToDevice,cudaMemcpyDeviceToHost
 */
#include "hip/hip_runtime.h"
 // helps to use threadId and blockId
//
#include <iostream>

__global__
void intro_threads(int *a,int *b, int *c) {
    c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}
//
void random_ints(int *a, int n) {
    for (int i = 0; i < n; ++i)
        a[i] = rand() % 100 + 1;
}
//
void display(int *a, int *b, int *result, int n) {
    for (int i = 0; i < n; i++) {
        printf("%d  +  %d  =  %d\n", a[i], b[i], result[i]);
    }
}
//
#define N 4
#define M 6
int main() {
    // the Host copies
    int *a , *b , *c ;
    // the Device copies
    int *d_a , *d_b , *d_c ;
    //size
    int size=N* sizeof(int);

    //allocating space for the Device copies
    hipMalloc((void **) &d_a,size);
    hipMalloc((void **) &d_b,size);
    hipMalloc((void **)&d_c,size);
    //allocate space for host copies of the variables, we dont accept values from a user
    //rather we randomly passing a number to each of them,
    a=(int *) malloc(size);
    random_ints(a,N);
    b=(int*) malloc(size);
    random_ints(b,N);
    c=(int *) malloc(size);


    //coping  to Device
    hipMemcpy(d_a , a , size , hipMemcpyHostToDevice);
    hipMemcpy(d_b , b , size , hipMemcpyHostToDevice);
    // launch the kernel
    intro_threads<<<1, N>>>(d_a, d_b, d_c ); // N threads for a single  block

    //coping results back to Host
    hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);
    //display the results
    display(a,b,c,N);

    //time to the cleanup stuff
        //for the Device variables
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    //for the Host variables
    free(a);
    free(b);
    free(c);
    return 0;
}
