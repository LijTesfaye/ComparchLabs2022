#include "hip/hip_runtime.h"

#include <iostream>

/*
Each parallel invocation of add_vector() is referred to as a 'block'.
 a set of blocks is called a grid.
if there is a questions saying how this  could be a parallel program, look how the addition of each
 * block is done independently.
 * ******
 * Block 0
 c[0]=a[0]+b[0]
 *********
 Block 1
 c[1]=a[1]+b[1]
 *
 e.t.c
 */
__global__
void add_vector(int *a, int *b, int *c){
    c[blockIdx.x]=a[blockIdx.x]+b[blockIdx.x];
    // bear in mind the usage of blockIdx, next example we substitute it with
    //threadId
}
//
void random_ints(int *a, int n) {
    for (int i = 0; i < n; ++i)
        a[i] = rand() % 100 + 1;
}
//
void display(int *a, int *b, int *result, int n) {
    for (int i = 0; i < n; i++) {
        printf("%d  + %d  =  %d\n", a[i], b[i], result[i]);
    }
}
//
//for the main
//
#define N 4
int main(void) {
    int *a , *b ,*c ; //host copies of a b c
    int *d_a, *d_b , *d_c; // device
    //size variable
    int size= N*sizeof(int);
    // allocate space for the device copies of a b c
    hipMalloc((void **) &d_a , size);
    hipMalloc((void **) &d_b , size);
    hipMalloc((void **) &d_c , size);
    //
    //get input values from the user, this part is a bit different from the previous two examples
    a=(int *) malloc(size);
    random_ints(a,N);
    b=(int *) malloc(size);
    random_ints(b,N);
    c=(int *)malloc(size);

    //copy the values to the Device
    hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
    hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);



    //call the kernel, N copies of add
    add_vector<<<N,1>>>(d_a,d_b,d_c); //N blocks with each one thread
    //copy the result to the Host
    hipMemcpy(c,d_c,size, hipMemcpyDeviceToHost);
    //display the results
    display(a,b,c,N);
    //cleanup
     //for cuda
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    //for c
    free(a);
    free(b);
    free(c);
    return 0;
}
